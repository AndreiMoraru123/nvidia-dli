
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>

__global__ void printNumber(int number) { printf("%d\n", number); }

int main() {
  hipStream_t streams[5];

  for (int i = 0; i < 5; ++i) {
    hipStreamCreate(&streams[i]);
    printNumber<<<1, 1, 0, streams[i]>>>(i);
    hipStreamDestroy(streams[i]);
  }
  hipDeviceSynchronize();
}
